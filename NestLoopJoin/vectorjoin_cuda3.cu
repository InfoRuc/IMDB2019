#include "hip/hip_runtime.h"
/* $begin mountainmain */
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>           /* gettimeofday */
#include <pthread.h>
#include <errno.h>
#include <hip/hip_runtime.h> 
#include <stdint.h>

#ifdef __cplusplus
extern "C" {
#endif

#if !defined(__i386__) && !defined(__x86_64__) && !defined(__sparc__)
#warning No supported architecture found -- timers will return junk.
#endif

static __inline__ uint64_t curtick() {
	uint64_t tick;
#if defined(__i386__)
	unsigned long lo, hi;
	__asm__ __volatile__ (".byte 0x0f, 0x31" : "=a" (lo), "=d" (hi));
	tick = (uint64_t) hi << 32 | lo;
#elif defined(__x86_64__)
	unsigned long lo, hi;
	__asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
	tick = (uint64_t) hi << 32 | lo;
#elif defined(__sparc__)
	__asm__ __volatile__ ("rd %%tick, %0" : "=r" (tick));
#endif
	return tick;
}

static __inline__ void startTimer(uint64_t* t) {
	*t = curtick();
}

static __inline__ void stopTimer(uint64_t* t) {
	*t = curtick() - *t;
}

#ifdef __cplusplus
}
#endif

#define L1 (1<<15)    /* Working set size for L1 cache 32KB */
#define L2 (1<<18)    /* Working set size for L2 cache 256KB */
#define L3 (1<<20)*2.5    /* Working set size for L3 cache 2.5MB */
#define LLC (1<<20)*55    /* Working set size for LLC cache 55MB */
#define MAXELEMS 600
#define random(x) (rand()%x)
#define nthread 4
#define nstream 4

#define THREAD_NUM 1024 //4096
#define BLOCK_NUM 13
#define DEVICE_MAX_COUNT 8

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

typedef struct {
  int *dev_data, *dev_vector, *results;
  int *dev_results[nstream];
  int vec_len, data_len;
  hipStream_t streams[nstream];
  uint64_t timer;
} device_args_t;

void init_data(int *data, int n, int cardinality);
double run_on_gpu(int *data, int n, int *vector, int vec_len, bool hasTransferTime);

void generator(sample_t *pk_table, int fk_len, sample_t *fk_table, int pk_len) {
  int i, j;
  for (i = 0; i < fk_len; i++) {
    fk_table[i].rowkey = i;
    fk_table[i].device_id = random(9);
    for (j = 0; j < 128; j++)
      fk_table[i].model[j] = 0;
  }
  for (i = 0; i < pk_len; i++){
    pk_table[i].rowkey = random(fk_len);
    pk_table[i].device_id = random(9);
    for (j = 0; j < 128; j++)
      pk_table[i].model[j] = 0;
  }
}


/* $begin mountainmain */
int main()
{
    int i, pk_len =L1/4*20/100, fk_len = MAXELEMS;

    sample_t* fk_table =(sample_t*)malloc(sizeof(sample_t) * fk_len);      /* foreign key column*/
    sample_t* pk_table =(sample_t*)malloc(sizeof(sample_t) * pk_len);   /* Initialize primary key vector elements in vector */

    generator(fk_table, fk_len, pk_table, pk_len);

    double time = run_on_gpu(fk_table, fk_len, pk_table, pk_len, false);
    printf("%4.2lf\n", time);
    
    return 0;
}

/* init_data - initializes the array */
void init_data(int *data, int n, int cardinality)
{
    int i;
    for (i = 0; i < n; i++)
        //data[i] = random(cardinality);
        data[i] = 1;
}

__global__ void cuda_nlj_thread(int *data, int *vector, int n, int vec_len, int *CountResult) {
  // used to save vector data
    __shared__ int shared[THREAD_NUM];
    __shared__ int rst_shared[THREAD_NUM];
  //  int result = 0;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    int vec_block_num = (vec_len + THREAD_NUM) / THREAD_NUM;
    //if (vec_block_num == 0) vec_block_num = 1;
    int vec_rem = vec_len;
    
    int j;
    int tid_limit = THREAD_NUM;

    rst_shared[tid] = 0;

    for (j = 0; j < vec_block_num; j++) {
      //if (j == (vec_block_num - 1) && vec_rem != 0)
        //tid_limit = vec_rem;
      tid_limit = (j == (vec_block_num -1)) ? vec_rem : THREAD_NUM;
      vec_rem -= THREAD_NUM;
      // copy vector to shared memory
      if (tid < tid_limit)
        shared[tid] = vector[THREAD_NUM * j + tid]; 

      __syncthreads();

      int i, k;
      // for debug
      int count = 0;
      for (i = bid * THREAD_NUM + tid; i < n; i += BLOCK_NUM * THREAD_NUM) {
        count++;
        for (k = 0; k < tid_limit; k++) {
          //if (data[i] == shared[k]) {
          if (1 == shared[k]) {
              rst_shared[tid] += 1;
          }
        }
      }
    }

    //__syncthreads();

    //rst_shared[tid] = result;

    __syncthreads();

    int i;
    if (tid == 0) {
        for (i = 1; i < THREAD_NUM; i++) {
            rst_shared[0] += rst_shared[i];
        }
        CountResult[bid] = rst_shared[0];
        
        //printf("count result[%d]: %d\n", bid, CountResult[bid]);
    }
}

double run_on_gpu(int *data, int n, int *vector, int vec_len, bool hasTransferTime) {
    int result = 0;
    //int *dev_data, *dev_vector;
    int *results;
    uint64_t timer1, timer2;
    double cycles_per_tuple1 = 0.0, cycles_per_tuple2 = 0.0;
    
    device_args_t deviceArgs[DEVICE_MAX_COUNT];
    int cudaDeviceNum = 0;
    // get number of device
    checkCuda(hipGetDeviceCount(&cudaDeviceNum));

    //hipStream_t streams[nstream];
    int i, j;

    int dataBlock = n / cudaDeviceNum; 
    int dataRemSize = n;
    int *tmpPtr = data;

    results = (int*)malloc(sizeof(int) * BLOCK_NUM);

    for (i = 0; i < cudaDeviceNum; i++) {
      int dataSize = (i == (cudaDeviceNum - 1) ? dataRemSize : dataBlock);
      checkCuda(hipSetDevice(i));

      dataRemSize -= dataSize;
      deviceArgs[i].results = (int*)malloc(sizeof(int) * BLOCK_NUM);
      checkCuda(hipMalloc((void**)&deviceArgs[i].dev_data, sizeof(int) * dataSize));
      checkCuda(hipMalloc((void**)&deviceArgs[i].dev_vector, sizeof(int) * vec_len));
    
      for (j = 0; j < nstream; j++) {
        checkCuda(hipMalloc((void**)&deviceArgs[i].dev_results[j], sizeof(int) * BLOCK_NUM));
      }
    }

    dataRemSize = n;
    hipStream_t copyStreams[DEVICE_MAX_COUNT];
    for (i = 0; i < cudaDeviceNum; i++) {
      int dataSize = (i == (cudaDeviceNum - 1) ? dataRemSize : dataBlock);
      checkCuda(hipSetDevice(i));

      dataRemSize -= dataSize;
      checkCuda(hipStreamCreate(&copyStreams[i]));
      checkCuda(hipMemcpyAsync(deviceArgs[i].dev_data, tmpPtr, sizeof(int)
            * dataSize, hipMemcpyHostToDevice, copyStreams[i])); 
      tmpPtr += dataSize;

      // Time for vector transfering and GPU computing
      startTimer(&timer2);
      checkCuda(hipMemcpyAsync(deviceArgs[i].dev_vector, vector, sizeof(int)
            * vec_len,  hipMemcpyHostToDevice, copyStreams[i])); 
      stopTimer(&timer2); 
    
      cycles_per_tuple1 += timer2;
    }

    dataRemSize = n;
    startTimer(&timer1);
    for (i = 0; i < cudaDeviceNum; i++) {
      int dataSize = (i == (cudaDeviceNum - 1) ? dataRemSize : dataBlock);
      checkCuda(hipSetDevice(i));

      int numDatastr = dataSize / nstream;
      int numData = dataSize;

      dataRemSize -= dataSize;
      
      checkCuda(hipStreamSynchronize(copyStreams[i]));  

      int tmpDataLen = 0;
      for (j = 0; j < nstream; j++) {
        tmpDataLen = (j == (nstream - 1) ? numData : numDatastr);
        printf("tmp data len:%d\n", tmpDataLen);
        numData -= numDatastr;
        checkCuda(hipStreamCreate(&deviceArgs[i].streams[j]));
        cuda_nlj_thread<<<BLOCK_NUM, THREAD_NUM, 2 * THREAD_NUM * sizeof(int),
          deviceArgs[i].streams[j]>>>(deviceArgs[i].dev_data + j * numDatastr,
              deviceArgs[i].dev_vector, tmpDataLen, vec_len, deviceArgs[i].dev_results[j]);
      }
    }

    for (i = 0; i < cudaDeviceNum; i++) {
      checkCuda(hipSetDevice(i));
      for (j = 0; j < nstream; j++) {
        checkCuda(hipStreamSynchronize(deviceArgs[i].streams[j]));
      }
    }
    
    stopTimer(&timer1); 
    cycles_per_tuple2 = (double)timer1 / (double)n;
    cycles_per_tuple1 = (double)cycles_per_tuple2 + ((double)cycles_per_tuple1 / (double)n);
     
    int k;
    for (i = 0; i < cudaDeviceNum; i++) {
      for (j = 0; j < nstream; j++) {
        checkCuda(hipMemcpy(results, deviceArgs[i].dev_results[j], sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost));
        for (k = 0; k < BLOCK_NUM; k++) {
            result += results[k];
        }
      }
    }

    for (i = 0; i < cudaDeviceNum; i++) {
      checkCuda(hipSetDevice(i));
      for (j = 0; j < nstream; j++) {
        checkCuda(hipStreamDestroy(deviceArgs[i].streams[j]));
        checkCuda(hipFree(deviceArgs[i].dev_results[j]));
      }
      checkCuda(hipFree(deviceArgs[i].dev_data));
      checkCuda(hipFree(deviceArgs[i].dev_vector));
    }

    printf("[%d]",result);
    return hasTransferTime ? cycles_per_tuple1 : cycles_per_tuple2;
}
